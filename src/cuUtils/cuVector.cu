/*
 * Image.cpp
 *
 *  Created on: 28 Aug 2016
 *      Author: morris
 */

#include <iostream>
#include <opencv2/core/matx.hpp>
#include <boost/timer/timer.hpp>

#include "cuVector.h"

using namespace std;

namespace cu {
    template<class T>
    Vector<T>::~Vector() {

    }

    template<class T>
    Vector<T>::Vector() {

    }

    template<class T>
    Vector<T>::Vector(const std::vector<T> &v) {
        size = v.size();
        CUDA_CHECK_RETURN(hipMalloc(&data, size * sizeof(T)));
        upload(v);
    }

    template<class T>
    void Vector<T>::upload(const std::vector<T> &v) {
        CUDA_CHECK_RETURN(hipMemcpy(data, v.data(), size * sizeof(T), hipMemcpyHostToDevice));
    }

    template<class T>
    Vector<T>::Vector(const std::size_t size) : size(size) {
        CUDA_CHECK_RETURN(hipMalloc(&data, size * sizeof(T)));
    }

    template<class T>
    std::vector<T> Vector<T>::download() const {
        std::vector<T> res(size);
        CUDA_CHECK_RETURN(hipMemcpy(&res[0], data, size * sizeof(T), hipMemcpyDeviceToHost));
        return res;
    }

    template class Vector<cv::Vec3b>;
}