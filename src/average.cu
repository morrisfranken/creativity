#include "hip/hip_runtime.h"
/*
 * average.cpp
 *
 *  Created on: 24 Mar 2018
 *      Author: morris
 */
#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <boost/timer/timer.hpp>

#include "average.h"
#include "GpuImage.h"
#include "utils.h"

using namespace std;
using namespace cv;

__global__ void scale_add(unsigned char *in, float *res, const size_t pitch_in, const size_t pitch_res, const size_t width_res, const size_t height_res, const float f_width, const float f_height) {
	const int row  = blockIdx.y * blockDim.y + threadIdx.y;
	const int col  = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < height_res & col < width_res) {	// make sure the thread is within the image
		const int idx_res = row * pitch_res + col * 3;
		const int idx_in = int(f_height * row) * pitch_in + int(f_width * col) * 3;

		for (int chan = 0; chan < 3; chan++)
			res[idx_res + chan] += (float)in[idx_in + chan];
	}
}

void average::run(const std::string &path) {
    const vector<string> files = my_utils::listdir(path);
	cv::Mat total_ = cv::Mat::zeros(1080/2, 1920/2, CV_32FC3);
	GpuImage gputotal(total_);

	constexpr int BLOCK_SIZE = 32;
	const dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	const dim3 dimGrid((gputotal.width + dimBlock.x - 1) / dimBlock.x, (gputotal.height + dimBlock.y - 1) / dimBlock.y);

    boost::timer::cpu_timer watch;
	#pragma omp parallel for num_threads(8) ordered schedule(dynamic)
	for (int i = 0; i < files.size(); i++) {
		const cv::Mat img = cv::imread(files[i], cv::IMREAD_COLOR);

//        cout << "load from " << omp_get_thread_num() << endl;
		#pragma omp ordered
		{
			if (i % 100 == 0)
				cout << i << " / " << files.size() << endl;

			GpuImage gpuimg(img);
			const float f_width = gpuimg.width / (float)gputotal.width;
			const float f_height = gpuimg.height / (float)gputotal.height;
			scale_add<<<dimGrid, dimBlock>>>(gpuimg.p(), (float *)gputotal.p(), gpuimg.pitch, gputotal.pitch / sizeof(float), gputotal.width, gputotal.height, f_width, f_height);

			if (omp_get_thread_num() == 0) {
				static int show_count = 0;
				show_count++;
				if (show_count%10==0) {
				Mat res = gputotal.download() / ((i+1) * 255.0f);
				imshow("frame", res);
				if (waitKey(1) == 27)
					exit(0);
				}
			}
		}
	}
    cout << "done in " << watch.format() << endl;

	Mat res = gputotal.download() / (double)(files.size());
    res.convertTo(res, CV_8UC3);
	imshow("frame", res);
	imwrite("average.png", res);
	waitKey();
}
